#include "hip/hip_runtime.h"
#include "render.cuh"
#include <assert.h>
#include <iomanip>
#include "rand.cuh"

struct split_by_completed {
    __host__ __device__ bool operator() (const PathState &segment) {
        return segment.remaining_iteration > 0;
    }
};

class BVH;

__global__ void compute_intersections(
    int total_path_count,
    bool is_empty,
    PathState *dev_path_state_buffer,
    HitRecord *dev_hit_record_buffer,
    DeviceBVH device_bvh) {
    int path_index = threadIdx.x + blockIdx.x * blockDim.x;
    if (path_index < total_path_count) {
        if (is_empty) return;
        PathState state = dev_path_state_buffer[path_index];
        cast_ray(state, dev_hit_record_buffer[path_index], device_bvh);
    }
    __syncthreads();
}

__global__ void gather(const unsigned int path_count, const PathState *path_state, float3 *image) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < path_count) {
        image[path_state[index].pixel_index] = image[path_state[index].pixel_index] + path_state[index].result;
    }
    __syncthreads();
}




__global__ void shade_material(const int path_count, PathState *path_state, HitRecord *records, Material *materials, const int iter, Aabb *lights, unsigned int light_count, DeviceBVH device_bvh) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < path_count && path_state[index].remaining_iteration > 0) {
        HitRecord hit_record = records[index];
        if (hit_record.missed) {
            path_state[index].attenuation = make_float3(0.0, 0.0, 0.0);
            path_state[index].remaining_iteration = 0;
        } else {
            Material material = materials[hit_record.material_index];
            thrust::default_random_engine rng;
            rng = make_seeded_random_engine(iter, index, path_state[index].remaining_iteration);
            material.shade(path_state[index], hit_record, rng, lights, light_count, device_bvh, materials);
        }
    }
    
    __syncthreads();
}

__global__ void generate_ray_from_camera(PathState *dev_path_state_buffer, Camera camera, int trace_depth, int iter) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < camera.pixel_horizontal_length && y < camera.pixel_vertical_length) {
        int index = x + y * camera.pixel_horizontal_length;
        PathState &state = dev_path_state_buffer[index];
        
        // calculat the ray params
        float x_offseted = x;
        float y_offseted = y;

        // anti_alias start
        thrust::default_random_engine rng = make_seeded_random_engine(iter, index, state.remaining_iteration);
        thrust::uniform_real_distribution<float> u_01(0.0f, 1.0f);
        x_offseted += u_01(rng);
        y_offseted += u_01(rng);
        // end
        const float u = (1.0f * x_offseted) / (float)camera.pixel_horizontal_length;
        const float v = (1.0f * y_offseted) / (float)camera.pixel_vertical_length;
        float3 o_offset = make_float3(0, 0, 0);
        if (camera.radius > 0.00001f) {
            o_offset = camera.radius * random_on_unit_disk(rng);
            o_offset = o_offset.x * unit(camera.horizontal) + o_offset.y * unit(camera.vertical);
        }
        state.ray.position = camera.origin + o_offset;
        state.ray.direction = unit(camera.lower_left_corner + u * camera.horizontal + v * camera.vertical - state.ray.position);
        state.ray.direction_inverse = 1.0f / state.ray.direction;
        // set other params
        state.attenuation = make_float3(1, 1, 1);
        state.result = make_float3(0, 0, 0);
        state.pixel_index = index;
        state.has_collect_direct_light = false;
        state.remaining_iteration = trace_depth;
    }
}

static void write_color(std::ostream &out, float3 pixel_color) {
    out << static_cast<int>(255.999 * clamp(0.00001f, 1, pixel_color.x)) << ' '
        << static_cast<int>(255.999 * clamp(0.00001f, 1, pixel_color.y)) << ' '
        << static_cast<int>(255.999 * clamp(0.00001f, 1, pixel_color.z)) << '\n';
}

static void show_progress_bar(int now, int total) {
    std::clog << '\r';
    std::clog << std::setw(6) << std::fixed << std::setprecision(2) << (float) now / total * 100.0f << "%";
}

void Render::path_trace() {
    using std::clog;
    dim3 threads_per_block2d(8, 8);
    const unsigned int pixel_count = camera.pixel_horizontal_length * camera.pixel_vertical_length;
    dim3 blocks_per_grid2d((camera.pixel_horizontal_length + threads_per_block2d.x - 1) / threads_per_block2d.x, (camera.pixel_vertical_length + threads_per_block2d.y - 1) / threads_per_block2d.y);
        
    int cur_depth = 0;
    int path_count = pixel_count;
    const unsigned int threads_per_block1d = 128;
    thrust::device_vector<float3> dev_image;
    thrust::host_vector<float3> host_image;
    dev_image.resize(pixel_count, make_float3(0, 0, 0));

    for (int i = 0; i < SPP; i++) {
        cur_depth = 0;
        generate_ray_from_camera<<<blocks_per_grid2d, threads_per_block2d>>>(thrust::raw_pointer_cast(dev_path_state_buffer.data()), camera, trace_depth, i);

        while (cur_depth < trace_depth) {
            dim3 blocks_per_grid1d((path_count + threads_per_block1d - 1) / threads_per_block1d);
            dev_hit_record_buffer.clear();
            dev_hit_record_buffer.resize(pixel_count, HitRecord());

            compute_intersections<<< blocks_per_grid1d, threads_per_block1d >>>(path_count, bvh.is_empty(), thrust::raw_pointer_cast(dev_path_state_buffer.data()), thrust::raw_pointer_cast(dev_hit_record_buffer.data()), bvh.get_dev_bvh());
            hipDeviceSynchronize();

            shade_material<<< blocks_per_grid1d, threads_per_block1d >>>(
                path_count, 
                thrust::raw_pointer_cast(dev_path_state_buffer.data()), 
                thrust::raw_pointer_cast(dev_hit_record_buffer.data()), 
                thrust::raw_pointer_cast(dev_material_buffer.data()), 
                i, 
                thrust::raw_pointer_cast(dev_direct_light_aabb_buffer.data()), 
                dev_direct_light_aabb_buffer.size(), 
                bvh.get_dev_bvh());

            auto pivot = thrust::partition(thrust::device, dev_path_state_buffer.begin(), dev_path_state_buffer.begin() + path_count, split_by_completed());
            path_count = pivot - dev_path_state_buffer.begin();
            if (path_count < 1) cur_depth = trace_depth;
            ++cur_depth;
        }
        path_count = pixel_count;
        dim3 blocks_per_grid1d((path_count + threads_per_block1d - 1) / threads_per_block1d);
        gather<<< blocks_per_grid1d, threads_per_block1d >>>(path_count, dev_path_state_buffer.data().get(), thrust::raw_pointer_cast(dev_image.data()));
        
        show_progress_bar(i, SPP);
    }
    host_image = dev_image;
    std::cout << "P3\n" << camera.pixel_horizontal_length << ' ' << camera.pixel_vertical_length << "\n255\n";
    for (int j = camera.pixel_vertical_length - 1; j >= 0 ; --j)
        for (int i = 0; i < camera.pixel_horizontal_length; ++i) {
            int index = i + j * camera.pixel_horizontal_length;
            write_color(std::cout, host_image[index] / SPP);
        }
}

void Render::init() {
    bvh.build();

    {
        thrust::host_vector<Sphere> host_sphere_temp = bvh.dev_spheres;
        thrust::host_vector<Triangle> host_triangle_temp = bvh.dev_triangles;

        for (const auto &sphere : host_sphere_temp) {
            if (host_material_buffer[sphere.material_index].emittance > 0.0f) {
                host_direct_light_aabb_buffer.push_back(sphere_aabb_getter()(sphere));
            }
        }

        for (const auto &triangle : host_triangle_temp) {
            if (host_material_buffer[triangle.material_index].emittance > 0.0f) {
                host_direct_light_aabb_buffer.push_back(triangle_aabb_getter()(triangle));
            }
        }
    }
    dev_direct_light_aabb_buffer = host_direct_light_aabb_buffer;

    dev_material_buffer = host_material_buffer;
    const int pixel_count = camera.pixel_vertical_length * camera.pixel_horizontal_length;
    host_path_state_buffer.resize(pixel_count);
    host_hit_record_buffer.resize(pixel_count);
    dev_path_state_buffer.resize(pixel_count);
    dev_hit_record_buffer.resize(pixel_count);
}




