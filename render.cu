#include "hip/hip_runtime.h"
#include "render.cuh"
#include <assert.h>
#include <iomanip>
#include "rand.cuh"

struct split_by_completed {
    __host__ __device__ bool operator() (const PathState &segment) {
        return segment.remaining_iteration > 0;
    }
};


class BVH;

__global__ void compute_intersections(
    int total_path_count,
    bool is_empty,
    PathState *dev_path_state_buffer,
    HitRecord *dev_hit_record_buffer,
    DeviceBVH device_bvh) {
    int path_index = threadIdx.x + blockIdx.x * blockDim.x;
    if (path_index < total_path_count) {
        if (is_empty) return;
        PathState state = dev_path_state_buffer[path_index];
        cast_ray(state, dev_hit_record_buffer[path_index], device_bvh);
    }
    __syncthreads();
}

__global__ void gather(const unsigned int path_count, const PathState *path_state, float3 *image) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < path_count) {
        image[path_state[index].pixel_index] = image[path_state[index].pixel_index] + path_state[index].result;
    }
    __syncthreads();
}




__global__ void shade_material(const int path_count, PathState *path_state, HitRecord *records, Material *materials, const int iter) {
    unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < path_count && path_state[index].remaining_iteration > 0) {
        HitRecord hit_record = records[index];
        if (hit_record.missed) {
            path_state[index].attenuation = make_float3(0.0, 0.0, 0.0);
            path_state[index].remaining_iteration = 0;
        } else {
            Material material = materials[hit_record.material_index];
            thrust::default_random_engine rng;
            rng = make_seeded_random_engine(iter, index, path_state[index].remaining_iteration);
            material.shade(path_state[index], hit_record, rng);
        }
    }
    
    __syncthreads();
}

__global__ void generate_ray_from_camera(PathState *dev_path_state_buffer, Camera camera, int trace_depth, int iter) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (x < camera.pixel_horizontal_length && y < camera.pixel_vertical_length) {
        int index = x + y * camera.pixel_horizontal_length;
        PathState &state = dev_path_state_buffer[index];
        
        // calculat the ray params
        float x_offseted = x;
        float y_offseted = y;

        // anti_alias start
        thrust::default_random_engine rng = make_seeded_random_engine(iter, index, state.remaining_iteration);
        thrust::uniform_real_distribution<float> u_01(0.0f, 1.0f);
        x_offseted += u_01(rng);
        y_offseted += u_01(rng);
        // end
        const float u = (1.0f * x_offseted) / (float)camera.pixel_horizontal_length;
        const float v = (1.0f * y_offseted) / (float)camera.pixel_vertical_length;
        float3 o_offset = make_float3(0, 0, 0);
        if (camera.radius > 0.00001f) {
            float3 o_offset = random_on_unit_sphere(rng);
            o_offset = camera.radius * (o_offset.x * unit(camera.horizontal) + o_offset.y * unit(camera.vertical));
        }
        state.ray.position = camera.origin + o_offset;
        state.ray.direction = unit(camera.lower_left_corner + u * camera.horizontal + v * camera.vertical - camera.origin - o_offset);
        state.ray.direction_inverse = 1.0f / state.ray.direction;
        // set other params
        state.attenuation = make_float3(1, 1, 1);
        state.result = make_float3(0, 0, 0);
        state.pixel_index = index;
        state.remaining_iteration = trace_depth;
    }
}

static void write_color(std::ostream &out, float3 pixel_color) {
    // assert(pixel_color.x <= 1.0f || pixel_color.y <= 1.0f || pixel_color.z <= 1.0f);
    // assert(pixel_color.x >= 0.0f || pixel_color.y >= 0.0f || pixel_color.z >= 0.0f);
    out << static_cast<int>(255.999 * clamp(0, 1, pixel_color.x)) << ' '
        << static_cast<int>(255.999 * clamp(0, 1, pixel_color.y)) << ' '
        << static_cast<int>(255.999 * clamp(0, 1, pixel_color.z)) << '\n';
}

static void show_progress_bar(int now, int total) {
    std::clog << '\r';
    std::clog << std::setw(6) << std::fixed << std::setprecision(2) << (float) now / total * 100.0f << "%";
}

void Render::path_trace() {
    using std::clog;
    dim3 threads_per_block2d(8, 8);
    const unsigned int pixel_count = camera.pixel_horizontal_length * camera.pixel_vertical_length;
    dim3 blocks_per_grid2d((camera.pixel_horizontal_length + threads_per_block2d.x - 1) / threads_per_block2d.x, (camera.pixel_vertical_length + threads_per_block2d.y - 1) / threads_per_block2d.y);
        
    int cur_depth = 0;
    int path_count = pixel_count;
    const unsigned int threads_per_block1d = 128;
    thrust::device_vector<float3> dev_image;
    thrust::host_vector<float3> host_image;
    dev_image.resize(pixel_count, make_float3(0, 0, 0));

    for (int i = 0; i < SPP; i++) {
        
        cur_depth = 0;
        generate_ray_from_camera<<<blocks_per_grid2d, threads_per_block2d>>>(thrust::raw_pointer_cast(dev_path_state_buffer.data()), camera, trace_depth, i);

        while (cur_depth < trace_depth) {
            dim3 blocks_per_grid1d((path_count + threads_per_block1d - 1) / threads_per_block1d);
            dev_hit_record_buffer.clear();
            dev_hit_record_buffer.resize(pixel_count, HitRecord());

            compute_intersections<<< blocks_per_grid1d, threads_per_block1d >>>(path_count, bvh.is_empty(), thrust::raw_pointer_cast(dev_path_state_buffer.data()), thrust::raw_pointer_cast(dev_hit_record_buffer.data()), bvh.get_dev_bvh());
            hipDeviceSynchronize();

            shade_material<<< blocks_per_grid1d, threads_per_block1d >>>(path_count, thrust::raw_pointer_cast(dev_path_state_buffer.data()), thrust::raw_pointer_cast(dev_hit_record_buffer.data()), thrust::raw_pointer_cast(dev_material_buffer.data()), i);
            auto pivot = thrust::partition(thrust::device, dev_path_state_buffer.begin(), dev_path_state_buffer.begin() + path_count, split_by_completed());
            path_count = pivot - dev_path_state_buffer.begin();
            if (path_count < 1) cur_depth = trace_depth;
            ++cur_depth;
        }
        path_count = pixel_count;
        dim3 blocks_per_grid1d((path_count + threads_per_block1d - 1) / threads_per_block1d);
        gather<<< blocks_per_grid1d, threads_per_block1d >>>(path_count, dev_path_state_buffer.data().get(), thrust::raw_pointer_cast(dev_image.data()));
        
        show_progress_bar(i, SPP);
    }
    host_image = dev_image;
    std::cout << "P3\n" << camera.pixel_horizontal_length << ' ' << camera.pixel_vertical_length << "\n255\n";
    for (int j = camera.pixel_vertical_length - 1; j >= 0 ; --j)
        for (int i = 0; i < camera.pixel_horizontal_length; ++i) {
            int index = i + j * camera.pixel_horizontal_length;
            write_color(std::cout, host_image[index] / SPP);
        }
    // print_image(thrust::raw_pointer_cast(host_image));
}

void Render::print_image(const float3 *image) {
    std::cout << "P3\n" << camera.pixel_horizontal_length << ' ' << camera.pixel_vertical_length << "\n255\n";
    for (int j = 0; j < camera.pixel_vertical_length; ++j)
        for (int i = 0; i < camera.pixel_horizontal_length; ++i) {
            int index = i + j * camera.pixel_horizontal_length;
            write_color(std::cout, host_path_state_buffer[index].attenuation / SPP);
        }
}

void Render::init() {
    bvh.build();
    bvh.host_nodes = bvh.dev_nodes;
    bvh.host_aabbs = bvh.dev_aabbs;
    // for (int i = 0; i < bvh.host_nodes.size(); ++i) {
    //     std::clog << "node " << i << '\n';
    //     std::clog << bvh.host_nodes[i].parent << ' ' << bvh.host_nodes[i].left_child << ' ' << bvh.host_nodes[i].right_child << ' ' << bvh.host_nodes[i].object_index << '\n';
    //     std::clog << bvh.host_aabbs[i].lower << ' ' << bvh.host_aabbs[i].upper << '\n';
    // }
    dev_material_buffer = host_material_buffer;
    const int pixel_count = camera.pixel_vertical_length * camera.pixel_horizontal_length;
    host_path_state_buffer.resize(pixel_count);
    host_hit_record_buffer.resize(pixel_count);
    dev_path_state_buffer.resize(pixel_count);
    dev_hit_record_buffer.resize(pixel_count);
    // dev_path_state_buffer.alloc(pixel_count * sizeof(PathState));
    // dev_hit_record_buffer.alloc(pixel_count * sizeof(HitRecord));
}




